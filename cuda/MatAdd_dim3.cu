#include <stdio.h>
#include <hip/hip_runtime.h>


// CUDA Kernel Device code
__global__ void MatAdd(float *A, float *B, float *C, int numCols, int numRows) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = j + i * numCols;
    if (i < numRows && j < numCols)
        C[idx] = A[idx] + B[idx];
}

// Host code
int main(void) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int numRows = 800;
    int numCols = 800;
    printf("Matrix addition of %d * %d elements\n", numRows, numCols);
    size_t size = numCols * numCols * sizeof(float);

    // 初始化输入矩阵A和B
    float A[numRows][numCols];
    float B[numRows][numCols];
    float C[numRows][numCols];
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            A[i][j] = rand() / (float)RAND_MAX;
            B[i][j] = rand() / (float)RAND_MAX;
            C[i][j] = A[i][j] + B[i][j];
        }
    }

    // Host内存分配
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            h_A[j + i * numCols] = A[i][j];
            h_B[j + i * numCols] = B[i][j];
        }
    }

    // Device内存分配
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将Host端的数据复制到Device端
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Host调用vectorAdd CUDA Kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (numRows * numCols + threadsPerBlock - 1) / threadsPerBlock;
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((numCols + threadsPerBlock.x - 1) / threadsPerBlock.x, (numRows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    MatAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numCols, numRows);

    // 将结果从Device端复制到Host端
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify that the result vector is correct
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            if (fabs(C[i][j] - h_C[j + i * numCols]) > 1e-5) {
                fprintf(stderr, "Result verification failed at element (%d, %d), truth is %f while get %f\n", i, j, C[i][j], h_C[j + i * numCols]);
                exit(EXIT_FAILURE);
            }
        }
    }

    // 释放Device内存和Host内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}