
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    hipDeviceProp_t prop;
    int count;
    err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to count cuda devices (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Number of CUDA devices: %d\n", count);
    for (int i = 0; i < count; ++i) {
        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to get cuda properties (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        printf("---General Information for device %d---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Total Global Memory: %ld\n", prop.totalGlobalMem);
        printf("Total Constant Memory: %ld\n", prop.totalConstMem);
        printf("Multiprocessor Count: %d\n", prop.multiProcessorCount);
        printf("Shared Memory per Multiprocessor: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per Multiprocessor: %d\n", prop.regsPerBlock);
        printf("Threads in Warp: %d\n", prop.warpSize);
        printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("Max Thread Dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max Grid Dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }

    int dev;
    hipGetDevice(&dev);  // 查看当前使用的CUDA设备
    printf("ID of current CUDA device: %d\n", dev);

    // 将某些属性填充到一个cudaDeviceProp结构中
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;

    hipChooseDevice(&dev, &prop);  // 返回满足条件的一个设备ID
    hipSetDevice(dev); // 设置CUDA设备
    
    
    int c;
    int *dev_c;
    size_t size = sizeof(int);
    err = hipMalloc((void **)&dev_c, size);  // 在设备上分配内存

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device number c (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    add<<<1, 1>>>(2, 7, dev_c);  // NVCC编译, <<<>>>中的参数告诉runtime如何启动设备代码, ()中为kernel函数参数

    err = hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy number c from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("2 + 7 = %d\n", c);

    printf("Hello World!\n");

    hipFree(dev_c);
}